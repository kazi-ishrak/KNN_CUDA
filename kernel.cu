#include "hip/hip_runtime.h"
﻿#include<stdio.h>
#include<iostream>
#include <hip/hip_runtime.h>
#include<fstream>
#include<string>
#include<math.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>

using namespace std;


__global__ void cal_distance(int i, int n, int* red, int* green, int* blue, int* test_red, int* test_green, int* test_blue, int *dis)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        float temp= (red[id] - test_red[i]) * (red[id] - test_red[i]) +
            (green[id] - test_green[i]) * (green[id] - test_green[i]) +
            (blue[id] - test_blue[i]) * (blue[id] - test_blue[i]);

        temp = sqrt(temp)+0.5;
       /* printf("%d ", int(temp));*/
        
        dis[id] = int(temp);
    }
}


void sorting(int n, int k, int* dis, int* res, char* cls) {
    thrust::device_vector<int> distances(dis, dis + n); // Copy distances to device vector
    thrust::device_vector<int> indices(n); // Device vector for indices
    thrust::sequence(thrust::device, indices.begin(), indices.end()); // Initialize indices
    thrust::sort_by_key(distances.begin(), distances.end(), indices.begin()); // Sort by distance

    // Extract the indices of the k nearest neighbors
    for (int x = 0; x < k; x++) {
        res[x] = indices[x];
    }
}

char vote(int k, int res[], char* cls)
{
    int r = 0, g = 0, b = 0, cnt = 1;
    for (int i = 0; i < k; i++)
    {
        
        char ch = cls[res[i]];
        
        if (ch == 'R')
            r++;
        else if (ch == 'G')
            g++;
        else
            b++;



        cnt++;
    }
    if (r == max(r, max(g, b)))
        return 'R';
    else if (g == max(r, max(g, b)))
        return 'G';
    else
        return 'B';

}



int main()
{
    int i, n;
    cout << "Enter the number of reference points:";
    cin >> n;
    cout << endl;

   //host pointers
    int* red, * green, * blue;
    char* cls;
    
   //cuda host mapped memory allocation
   hipError_t status_red= hipHostAlloc((void**)&red, n * sizeof(int), hipHostMallocMapped);
   hipError_t status_green = hipHostAlloc((void**)&green, n * sizeof(int), hipHostMallocMapped);
   hipError_t status_blue = hipHostAlloc((void**)&blue, n * sizeof(int), hipHostMallocMapped);
   hipError_t status_cls = hipHostAlloc((void**)&cls, n * sizeof(char), hipHostMallocMapped);

   //check allocation error
   if (status_red != hipSuccess)
   {
       cout << "Host allocation failed: " <<hipGetErrorString(status_red)<< endl;
       return 1;

   }
   if (status_green != hipSuccess)
   {
       cout << "Host allocation failed: " << hipGetErrorString(status_green) << endl;
       return 1;
   }
   if (status_blue != hipSuccess)
   {
       cout << "Host allocation failed: " << hipGetErrorString(status_blue) << endl;
       return 1;
   }
   if (status_cls != hipSuccess)
   {
       cout << "Host allocation failed: " << hipGetErrorString(status_cls) << endl;
       return 1;
   }
   
   //obtain device pointer
   int* d_red, *d_green, *d_blue;
   char *d_cls;
   hipHostGetDevicePointer(&d_red, red, 0);
   hipHostGetDevicePointer(&d_green, green, 0);
   hipHostGetDevicePointer(&d_blue, blue, 0);
   hipHostGetDevicePointer(&d_cls, cls, 0);


   ifstream file;
   file.open("final_data.txt");

   if (!file.is_open())
   {
       cout << "Error opening file" << std::endl;
   }
   for (i = 0; i < n * 4; i = i + 4)
   {
       file >> red[i / 4];
       file >> green[i / 4];
       file >> blue[i / 4];
       file >> cls[i / 4];
   }
   file.close();

   int* test_red, * test_green, * test_blue;
   char* test_cls;


   //cuda host mapped memory allocation
   hipError_t status_test_red = hipHostAlloc((void**)&test_red, n * sizeof(int), hipHostMallocMapped);
   hipError_t status_test_green = hipHostAlloc((void**)&test_green, n * sizeof(int), hipHostMallocMapped);
   hipError_t status_test_blue = hipHostAlloc((void**)&test_blue, n * sizeof(int), hipHostMallocMapped);
   hipError_t status_test_cls = hipHostAlloc((void**)&test_cls, n * sizeof(char), hipHostMallocMapped);

   //check allocation error
   if (status_test_red != hipSuccess)
   {
       cout << "Host allocation failed: " << hipGetErrorString(status_test_red) << endl;
       return 1;

   }
   if (status_test_green != hipSuccess)
   {
       cout << "Host allocation failed: " << hipGetErrorString(status_test_green) << endl;
       return 1;
   }
   if (status_test_blue != hipSuccess)
   {
       cout << "Host allocation failed: " << hipGetErrorString(status_test_blue) << endl;
       return 1;
   }
   if (status_test_cls != hipSuccess)
   {
       cout << "Host allocation failed: " << hipGetErrorString(status_test_cls) << endl;
       return 1;
   }


   //obtain device pointer
   int* d_test_red, * d_test_green, * d_test_blue;
   char* d_test_cls;
   hipHostGetDevicePointer(&d_test_red, test_red, 0);
   hipHostGetDevicePointer(&d_test_green, test_green, 0);
   hipHostGetDevicePointer(&d_test_blue, test_blue, 0);
   hipHostGetDevicePointer(&d_test_cls, test_cls, 0);



   ifstream file2;
   file2.open("test_data.txt");
   int m;
   cout << "Enter the number of test points:";
   cin >> m;
   for (i = 0; i < m * 4; i = i + 4)
   {
       file2 >> test_red[i / 4];
       file2 >> test_green[i / 4];
       file2 >> test_blue[i / 4];
       file2 >> test_cls[i / 4];
   }
   file2.close();
   int k;
   cout << "Choose your value of k:";
   cin >> k;

   for (i = 0; i < m; i++)
   {
       int threadsPerBlock = 256;
       int blocksPerGrid = (n + 255) / 256;

       int *dis;
       hipError_t status_dis = hipHostAlloc((void**)&dis, n * sizeof(int), hipHostMallocMapped);
       if (status_dis != hipSuccess)
       {
           cout << "Host allocation failed: " << hipGetErrorString(status_dis) << endl;
           return 1;

       }

       int* d_dis;
       hipHostGetDevicePointer(&d_dis, dis, 0);
       cal_distance << <blocksPerGrid,threadsPerBlock >> > (i, n, d_red, d_green, d_blue, d_test_red, d_test_green, d_test_blue, d_dis);
       hipDeviceSynchronize();

       hipError_t error1 = hipGetLastError();
       if (error1 != hipSuccess) {
           printf("CUDA error: %s\n", hipGetErrorString(error1));
       }





       int* res;
       hipError_t status_res = hipHostAlloc((void**)&res, k * sizeof(int), hipHostMallocMapped);
       if (status_res != hipSuccess)
       {
           cout << "Host allocation failed: " << hipGetErrorString(status_res) << endl;
           return 1;

       }
   
       sorting(n, k, dis, res, cls);

       thrust::device_vector<int> distances(dis, dis + n);
       thrust::device_vector<int> indices(n);
       thrust::sequence(thrust::device, indices.begin(), indices.end());
       thrust::sort_by_key(distances.begin(), distances.end(), indices.begin());

       //now vote
       
       cout << vote(k, res, cls) << endl;

       hipHostFree(dis);
       hipHostFree(res);
   }

   hipHostFree(red);
   hipHostFree(green);
   hipHostFree(blue);
   hipHostFree(cls);

   hipHostFree(test_red);
   hipHostFree(test_green);
   hipHostFree(test_blue);
   hipHostFree(test_cls);
}